#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"
#include "math_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>


typedef struct Orb {
	double x;
	double y;
	double z;
	double vx;
	double vy;
	double vz;
	double m;
	double st;
}Orb;

#define G 0.000005
#define MIN_DIST 1.0
const int unitSize = 8;//unitSize:size of each unit:8 floats
double wide = 10000;
double mass = 10;
double velo = 0.005;
int saveTimes = 0;
int calcTimes = 1000;


hipError_t calcOrbsWithCuda(Orb* olist, int nUnit, int nTimes, const char* saveFile);

//__global__ void calcOne(Orb*o, int oId, Orb*olist, int nUnit);
__global__ void calcOneInDevice(void *polist, int nUnit);
Orb* newOrbList(int nUnit, int style);
void deleteOrbList(Orb *olist);
void initOrbList(Orb *olist, int nUnit, int style);
void printList(Orb* olist, int nNum);
void saveList(Orb* olist, int nNum, const char* saveFile);
#define PRINTLIST(str,o) printf("%s{%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf}\n", str, (o)->x, (o)->y, (o)->z, (o)->vx, (o)->vy, (o)->vz, (o)->m, (o)->st)

__global__ void calcOneInDevice(void *polist, int nUnit)
{
	int tid = threadIdx.x;
	//c[i] = a[i] + b[i];
	printf("this is thread:%d,%d,%d grid:%d,%d,%d blockIdx:%d,%d,%d nUnit=%d\n", threadIdx.x, threadIdx.y, threadIdx.z, gridDim.x, gridDim.y, gridDim.z, blockIdx.x, blockIdx.y, blockDim.z, nUnit);
	Orb* olist = (Orb*)polist;
	Orb* o = olist + tid;
	int oId = tid;
	//if (o->st < 0) {
	int i = 0, isTooRappid = 0;
	double gax = 0, gay = 0, gaz = 0, dist = 0;
	printf("o[%d]={%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf}\n", tid, (o)->x, (o)->y, (o)->z, (o)->vx, (o)->vy, (o)->vz, (o)->m, (o)->st);
	
	for (i = 0; i<nUnit; ++i) {
		Orb* ta = olist + i;
		printf("ta[%d]={%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf}\n", tid, (ta)->x, (ta)->y, (ta)->z, (ta)->vx, (ta)->vy, (ta)->vz, (ta)->m, (ta)->st);

		if (o->st < 0 && ta->st < 0 && oId != i) {
			dist = sqrt(((ta->x - o->x)*(ta->x - o->x) + (ta->y - o->y)*(ta->y - o->y) + (ta->z - o->z)*(ta->z - o->z)));
			isTooRappid = dist*dist<(o->vx*o->vx + o->vy*o->vy + o->vz*o->vz) * 10;
			if (dist<MIN_DIST || isTooRappid) {
				// crash
				if (o->m < ta->m) {
					o->st = -o->st;
					printf("one crash: oid=%d, tid=%d dist=%f isTooRappid=%d\n", oId, i, dist, isTooRappid);
				}
				continue;
			}
			//calcGravity<<<1,1>>>(o, ta, dist, &gax, &gay, &gaz);

			double a = ta->m / (dist*dist) * G;
			gax += -a * (o->x - ta->x) / dist;
			gay += -a * (o->y - ta->y) / dist;
			gaz += -a * (o->z - ta->z) / dist;
		}
	}
	o->x += o->vx;
	o->y += o->vy;
	o->z += o->vz;
	o->vx += gax;
	o->vy += gay;
	o->vz += gaz;
	//}
}

Orb* newOrbList(int nUnit, int style) {

	int listSizeOfByte = sizeof(double) * unitSize * nUnit;
	Orb* list = (Orb*)malloc(listSizeOfByte);
	initOrbList(list, nUnit, style);
	return list;
}

void deleteOrbList(Orb *olist) {
	if (olist != NULL) {
		delete(olist);
	}
}
void initOrbList(Orb *olist, int nUnit, int style) {
	int i = 0;
	srand(time(NULL));
	for (i = 0; i<nUnit; ++i) {
		//Orb* o = (Orb*)(list+i*unitSize);
		Orb* o = olist + i;
		o->x = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->y = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->z = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->vx = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->vy = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->vz = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->m = (double)rand() / (double)RAND_MAX*mass;
		o->st = -(double)i;
	}
}

int getArgInt(const int argc, const char** argv, const char* flag, int defaultValue) {
	int value = defaultValue;
	for (int i = 0; i < argc; i++) {
		if (0 == strcmp(flag, argv[i]) && i<argc-1) {
			sscanf(argv[i + 1], "%d", &value);
			break;
		}
	}
	return value;
}
const char* getArgStr(const int argc, const char** argv, const char* flag, const char* defaultValue) {
	const char* value = defaultValue;
	for (int i = 0; i < argc; i++) {
		if (0 == strcmp(flag, argv[i]) && i<argc - 1) {
			value = argv[i + 1];
			break;
		}
	}
	return value;
}



int main(int argc, const char**argv)
{
int nUnit = getArgInt(argc, argv, "-n", 100);
int nTimes = getArgInt(argc, argv, "-t", 1000);
const char* saveFile = getArgStr(argc, argv, "--savefile", "thelist1");

printf("nUnit=%d, nTimes=%d\n", nUnit, nTimes);

hipError_t cudaStatus;
Orb* olist = newOrbList(nUnit, 1);

printList(olist, nUnit);

cudaStatus = calcOrbsWithCuda(olist, nUnit, nTimes, saveFile);
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "calcOrbsWithCuda failed!");
	return 1;
}



// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipDeviceReset failed!");
	return 1;
}

printf("calc done-------------------\n");
printList(olist, nUnit);

saveList(olist, nUnit, saveFile);
printf("save ok %s -------------------\n", saveFile);

deleteOrbList(olist);
scanf("%d", &nUnit);


return 0;
}

hipError_t calcOrbsWithCuda(Orb* olist, int nUnit, int nTimes, const char* saveFile) {
	Orb* dev_a = 0;
	//int* dev_b = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_a, nUnit * sizeof(Orb));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dev_a, (void*)olist, nUnit * sizeof(Orb), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyHostToDevice failed!");
		goto Error;
	}

	for (int i = 0; i < nTimes; ++i) {

		// do
		calcOneInDevice << <1, nUnit, nUnit*sizeof(Orb), 0 >> >(dev_a, nUnit);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calcOneInDevice launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		if (i*nUnit*nUnit > 10000) {
			hipMemcpy((void*)olist, dev_a, nUnit * sizeof(Orb), hipMemcpyDeviceToHost);
			saveList(olist, nUnit, saveFile);
		}
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy((void*)olist, dev_a, nUnit * sizeof(Orb), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	//hipFree(dev_b);

	return cudaStatus;
}


void printList(Orb* olist, int nNum) {
	for (int i = 0; i < nNum; i++) {
		printf("{%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf}\n", (olist + i)->x, (olist + i)->y, (olist + i)->z, (olist + i)->vx, (olist + i)->vy, (olist + i)->vz, (olist + i)->m, (olist + i)->st);
	}
}

void saveList(Orb* olist, int nNum, const char* saveFile) {
	if (saveFile != NULL) {
		FILE* fhandle = fopen(saveFile, "w");

		if (olist != NULL) {
			fprintf(fhandle, "[");
			for (int i = 0; i < nNum; i++) {
				fprintf(fhandle, "{%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf},", (olist + i)->x, (olist + i)->y, (olist + i)->z, (olist + i)->vx, (olist + i)->vy, (olist + i)->vz, (olist + i)->m, (olist + i)->st);
			}
			fseek(fhandle, -1, SEEK_CUR);
			fprintf(fhandle, "]");
		}

		fclose(fhandle);
	}
}