
#include "hip/hip_runtime.h"

#include "hip/hip_math_constants.h"
#include "math_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

typedef struct Orb{
	double x;
	double y;
	double z;
	double vx;
	double vy;
	double vz;
	double m;
	double st;
}Orb;

#define G 0.000005
#define MIN_DIST 1.0
const int unitSize = 8;//unitSize:ÿ�����㵥λ�Ĵ�С�����ٸ�float
double wide = 10000;
double mass = 10;
double velo = 0.005;
int saveTimes = 0;

void calcGravity(Orb*o, Orb*ta, double dist, double*gx, double*gy, double*gz);
void calcOne(Orb*o, int oId, Orb*olist, int nUnit);
Orb* newOrbList(int nUnit, int style);
void deleteOrbList(Orb *olist);
void initOrbList(Orb *olist, int nUnit, int style);

/* calc gravity between two*/
void calcGravity(Orb*o, Orb*ta, double dist, double*gx, double*gy, double*gz) {
	double a = ta->m / (dist*dist) * G;
	*gx += -a * (o->x - ta->x) / dist;
	*gy += -a * (o->y - ta->y) / dist;
	*gz += -a * (o->z - ta->z) / dist;
}
/* Orb update once with list */
void calcOne(Orb*o, int oId, Orb*olist, int nUnit) {
	//if (o->st < 0) {
	int i = 0, isTooRappid = 0;
	double gax = 0, gay = 0, gaz = 0, dist = 0;
	for (i = 0; i<nUnit; ++i) {
		Orb* ta = olist + i;
		if (o->st < 0 && ta->st < 0 && oId != i) {
			dist = sqrt(((ta->x - o->x)*(ta->x - o->x) + (ta->y - o->y)*(ta->y - o->y) + (ta->z - o->z)*(ta->z - o->z)));
			isTooRappid = dist*dist<(o->vx*o->vx + o->vy*o->vy + o->vz*o->vz) * 10;
			if (dist<MIN_DIST || isTooRappid) {
				// crash
				if (o->m < ta->m) {
					o->st = -o->st;
					printf("one crash: oid=%d, tid=%d dist=%f isTooRappid=%d\n", oId, i, dist, isTooRappid);
				}
				continue;
			}
			calcGravity(o, ta, dist, &gax, &gay, &gaz);
		}
	}
	o->x += o->vx;
	o->y += o->vy;
	o->z += o->vz;
	o->vx += gax;
	o->vy += gay;
	o->vz += gaz;
	//}
}

Orb* newOrbList(int nUnit, int style) {

	int listSizeOfByte = sizeof(double) * unitSize * nUnit;
	Orb* list = (Orb*)malloc(listSizeOfByte);
	initOrbList(list, nUnit, style);
	return list;
}

void deleteOrbList(Orb *olist) {
	if (olist != NULL) {
		delete(olist);
	}
}
void initOrbList(Orb *olist, int nUnit, int style) {
	int i = 0;
	srand(time(NULL));
	for (i = 0; i<nUnit; ++i) {
		//Orb* o = (Orb*)(list+i*unitSize);
		Orb* o = olist + i;
		o->x = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->y = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->z = (double)rand() / (double)RAND_MAX*wide - wide / 2.0;
		o->vx = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->vy = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->vz = (double)rand() / (double)RAND_MAX*velo - velo / 2.0;
		o->m = (double)rand() / (double)RAND_MAX*mass;
		o->st = -(double)i;
	}
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t calcOrbsWithCuda(Orb* olist, int nUnit, int nTimes) {
	int* dev_a = 0;
	int* dev_b = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_a, nUnit * sizeof(Orb));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
